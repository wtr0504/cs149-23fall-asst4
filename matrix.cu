#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

__global__ void hello_from_gpu()
{
    printf("Hello World from the the GPU\n");
}

__global__ void matrix_mul(float* x, float * y, float* z, int m, int n, int l)
{
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int row = idx / m;
  const int col = idx % m;
  
  if(row < m && col < n) {
    for(int i = 0; i < l; i++) {
      z[row*n+ col] += x[row*l + i] * y[i*n + col];

    }
    // if(row*n + col < 10){
    //   printf("z[%d]:%f\t\n",row*n + col,z[row*n+ col]);
    // }
  }
}

__global__ void matrix_mul_1(float* x, float * y, float* z, int m, int n, int l)
{
  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  // for(; bidx < m; bidx += gridDim.x)
  {
    for(;tidx < n; tidx += blockDim.x) {
      for(int i = 0; i < l; i++) {
        z[bidx*n + tidx] += x[bidx*l + i] * y[i*n + tidx];
      }
    }
  }
}

__global__ void matrix_mul_2(float* x, float * y, float* z, int m, int n, int l)
{
  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  extern __shared__ float data[];
  for(int i = tidx; i < l; i += blockDim.x) {
    data[i] = x[bidx*l + i];
  }

  // 注意调用这个函数保证该 block 里面所有的线程同步， 
  // 因为该 block 里面所有的线程需要协同工作，一起将 m*l 矩阵中的第 bidx 行的元素写入 data 中。
  __syncthreads();

  // for(; bidx < m; bidx += gridDim.x)
  {
    for(;tidx < n; tidx += blockDim.x) {
      for(int i = 0; i < l; i++) {
        z[bidx*n + tidx] += data[i] * y[i*n + tidx];
      }
    }
  }
}



void matrix_mul_cuda(float* x, float * y, float* z, int m, int n, int l){
  if(x == NULL || y == NULL || z == NULL)
    return ;
  float *d_x, *d_y, *d_z;
  // std::cout<<"cuda X: ";

  // for(int i = 0; i < 10; i++) {
  //   std::cout << x[i] << " ";
  // }
  // std::cout<<""<<std::endl;
  
  hipMalloc((void**)&d_x, m*l*sizeof(float));
  hipMalloc((void**)&d_y, l*n*sizeof(float));
  hipMalloc((void**)&d_z, m*n*sizeof(float));

  // 将host数据拷贝到device
  hipMemcpy((void*)d_x, (void*)x, m*l*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void*)d_y, (void*)y, l*n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void*)d_z, (void*)z, m*n*sizeof(float), hipMemcpyHostToDevice);
  // 定义kernel的执行配置
  // dim3 threads(1024);
  // dim3 blocks(m*n-1024+1/1024);
  // matrix_mul <<<blocks, threads>>>(d_x, d_y, d_z, m, n, l);

  dim3 blocks(m);
  dim3 threads(1024);
  matrix_mul_1 <<<blocks, threads>>>(d_x, d_y, d_z, m, n, l);

  // 将device得到的结果拷贝到host
  hipMemcpy((void*)z, (void*)d_z, m*n*sizeof(float), hipMemcpyDeviceToHost);

  // 释放device内存
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  // std::cout<<"cuda : ";
  // for(int i = 0; i < 10; i++) {
  //   std::cout << z[i] << " ";
  // }
  // std::cout<<""<<std::endl;
}

int main()
{
  int M = 2048;
  int L = 1024;
  int N = 512;

  // hello_from_gpu<<<4, 4>>>();
  // cudaDeviceSynchronize();
  // 申请host内存
  float *x = NULL;
  float *y = NULL;
  float *z = NULL;
  x = (float*)malloc(M*L*sizeof(float));
  y = (float*)malloc(L*N*sizeof(float));
  z = (float*)malloc(M*N*sizeof(float));

  if(x == NULL || y == NULL || z == NULL)
    return 0;
  
  // 初始化数据
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < L; ++j) {
      x[i*L + j] = 1.1;
    }
  }
  for (int i = 0; i < L; ++i) {
    for (int j = 0; j < N; ++j) {
      y[i*N + j] = 1.1;
    }
  }
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      z[i*N + j] = 0;
    }
  }

  // 申请device内存
  float *d_x, *d_y, *d_z;
  hipMalloc((void**)&d_x, M*L*sizeof(float));
  hipMalloc((void**)&d_y, L*N*sizeof(float));
  hipMalloc((void**)&d_z, M*N*sizeof(float));

  // 将host数据拷贝到device
  hipMemcpy((void*)d_x, (void*)x, M*L*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void*)d_y, (void*)y, L*N*sizeof(float), hipMemcpyHostToDevice);

  // 定义kernel的执行配置
  dim3 threads(1024);
  dim3 blocks(M*N-1024+1/1024);
  matrix_mul <<<blocks, threads>>>(d_x, d_y, d_z, M, N, L);

  // 将device得到的结果拷贝到host
  hipMemcpy((void*)z, (void*)d_z, M*N*sizeof(float), hipMemcpyDeviceToHost);

  // 输出前10个数值
  for(int i = 0; i < 10; i++) {
    std::cout << z[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "Done!" << std::endl;

  // 释放device内存
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  // 释放host内存
  free(x);
  free(y);
  free(z);

  return 0;
}
